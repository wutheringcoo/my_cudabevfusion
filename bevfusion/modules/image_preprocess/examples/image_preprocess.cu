#include "hip/hip_runtime.h"
/*
input:

Todolists:
1) C++ Implementation Of Nearest Interpolation/ Cuda Implementation/ Python Implementation
2)  Consistency verification  above
*/
#define cuda_2d_launch(kernel, stream, nx, ny, nz, ...)                          \
    do {                                                                         \
        dim3 __threads__(32, 32);                                                \
        dim3 __blocks__(divup(nx, 32), divup(ny, 32), nz);                       \
        kernel<<<__blocks__, __threads__, 0, stream>>>(nx, ny, nz, __VA_ARGS__); \
        nv::check_runtime(hipPeekAtLastError(), #kernel, __LINE__, __FILE__);   \
    } while (false)
#endif  // CUDA_DEBUG
}
;       // namespace nv#define cuda_2d_launch(kernel, stream, nx, ny, nz, ...)                      \
  do {                                                                       \
    dim3 __threads__(32, 32);                                                \
    dim3 __blocks__(divup(nx, 32), divup(ny, 32), nz);                       \
    kernel<<<__blocks__, __threads__, 0, stream>>>(nx, ny, nz, __VA_ARGS__); \
    nv::check_runtime(hipPeekAtLastError(), #kernel, __LINE__, __FILE__);   \
  } while (false)
#endif  // CUDA_DEBUG
}
;  // namespace nv

cuda_2d_launch(
    normalize_to_planar_kernel_function,
    _stream,
    param_.output_width,
    param_.output_height,
    param_.num_camera,
    sx_,
    sy_,
    crop_x_,
    crop_y_,
    reinterpret_cast<uchar3*>(raw_images_),
    param_.image_width,
    param_.image_height,
    normalize_images_,
    param_.method);

template <NormType norm_type, Interpolation interpolation, typename OutputType>
static __global__ void normalize_to_planar_kernel(
    int nx,
    int ny,
    int nz,
    float sx,
    float sy,
    int crop_x,
    int crop_y,
    uchar3* imgs,
    int image_width,
    int image_height,
    void* output,
    NormMethod method) {
    int ix = cuda_2d_x;
    int iy = cuda_2d_y;
    if (ix >= nx || iy >= ny)
        return;

    int icamera = blockIdx.z;
    uchar3* img = imgs + image_width * image_height * icamera;
    uchar3 pixel = load_pixel<interpolation>(img, ix, iy, crop_x, crop_y, sx, sy, image_width, image_height);

    if (method.channel_type == ChannelType::Invert) {
        unsigned char t = pixel.z;
        pixel.z = pixel.x;
        pixel.x = t;
    }

    template <>
    __device__ uchar3 load_pixel<Interpolation::Nearest>(
        const uchar3* image,  // src_image
        int x,                // dst_x
        int y,
        int tox,
        int toy,
        float sx,
        float sy,
        int width,
        int height) {
        // In some cases, the floating point precision will lead to miscalculation of the value,
        // making the result not exactly match with opencv,
        // so here you need to add eps as precision compensation
        //
        // A special case is when the input is 3840 and the output is 446, x = 223:
        // const int src_x_double = 223.0  * (3840.0  / 446.0);            // -> 1920
        // const int src_x_float  = 223.0f * (3840.0f / 446.0f);           // -> 1919
        // const int src_x_float  = 223.0f * (3840.0f / 446.0f) + 1e-5;    // -> 1920
        //
        // !!! If you want to use the double for sx/sy, you'll get a 2x speed drop
        const float eps = 1e-5;
        int ix = (x + tox) * sx + eps;
        int iy = (y + toy) * sy + eps;
        return image[iy * width + ix];
    }