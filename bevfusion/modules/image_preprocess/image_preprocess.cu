#include "hip/hip_runtime.h"
/*
input:

Todolists:
1) C++ Implementation Of Nearest Interpolation/ Cuda Implementation/ Python Implementation
2)  Consistency verification  above
*/

template <>
__device__ uchar3 load_pixel<Interpolation::Nearest>(
    const uchar3* image,  // src_image
    int x,                // dst_x
    int y,
    int tox,
    int toy,
    float sx,
    float sy,
    int width,
    int height) {
    // In some cases, the floating point precision will lead to miscalculation of the value,
    // making the result not exactly match with opencv,
    // so here you need to add eps as precision compensation
    //
    // A special case is when the input is 3840 and the output is 446, x = 223:
    // const int src_x_double = 223.0  * (3840.0  / 446.0);            // -> 1920
    // const int src_x_float  = 223.0f * (3840.0f / 446.0f);           // -> 1919
    // const int src_x_float  = 223.0f * (3840.0f / 446.0f) + 1e-5;    // -> 1920
    //
    // !!! If you want to use the double for sx/sy, you'll get a 2x speed drop
    const float eps = 1e-5;
    int ix = (x + tox) * sx + eps;
    int iy = (y + toy) * sy + eps;
    return image[iy * width + ix];
}
