#include <hip/hip_runtime.h>

#include <stdio.h>
// #include <ATen/cuda/CUDAContext.h>
#define THREADS_BLOCK_X   3
#define THREADS_BLOCK_Y   3
#define THREADS_PER_BLOCK THREADS_BLOCK_X *THREADS_BLOCK_Y
#define DIVUP(m, n)       ((m) / (n) + ((m) % (n) > 0))

static __global__ void nearest_neighbor_interpolate_kernel(
    int height_dist,
    int width_dist,
    unsigned char *image_dst,
    float scale_x,
    float scale_y,
    int heigth_src,
    int width_src,
    const unsigned char *image_src) {
    int ix_dist = blockIdx.x * blockDim.x + threadIdx.x;
    int iy_dist = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix_dist >= width_dist || iy_dist >= height_dist)
        return;

    // const float eps = 1e-5;
    int ix_src = ix_dist / scale_x;
    int iy_src = iy_dist / scale_y;
    printf(
        "%d, %d, %f, %d,  %d\n",
        ix_src,
        ix_dist,
        scale_x,
        image_src[iy_src * width_src + ix_src],
        iy_dist * width_dist + ix_dist);

    image_dst[iy_dist * width_dist + ix_dist] = image_src[iy_src * width_src + ix_src];
    printf("*%d, *%d\n", iy_dist * width_dist + ix_dist, image_dst[iy_dist * width_dist + ix_dist]);
}

void nearest_neighbor_interpolate_kernel_launcher(
    hipStream_t stream,
    int height_dist,
    int width_dist,
    unsigned char *image_dst,
    float scale_x,
    float scale_y,
    int heigth_src,
    int width_src,
    unsigned char *image_src) {
    hipError_t err;

    dim3 threads(THREADS_BLOCK_X, THREADS_BLOCK_Y);
    dim3 blocks(DIVUP(width_dist, THREADS_BLOCK_X), DIVUP(height_dist, THREADS_BLOCK_Y));

    nearest_neighbor_interpolate_kernel<<<blocks, threads, 0, stream>>>(
        height_dist, width_dist, image_dst, scale_x, scale_y, heigth_src, width_src, image_src);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

int main() {
    // cudaStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    hipStream_t stream;
    hipStreamCreate(&stream);

    int height_dist = 3;  // dst
    int width_dist = 3;
    unsigned char *img_dst_cuda;
    hipError_t cudaStatus = hipMalloc((void **)&img_dst_cuda, height_dist * width_dist * 1 * sizeof(unsigned char *));
    if (cudaStatus != hipSuccess) {
        printf("img dst CudaMalloc failed!");
    }

    int heigth_src = 4;
    int width_src = 4;
    unsigned char img_arr[] = {20, 50, 80, 160, 110, 140, 170, 244, 200, 230, 255, 215, 1, 2, 3, 4};
    unsigned char *img_src = img_arr;
    unsigned char *img_src_cuda;

    float scale_x = static_cast<float>(width_dist) / width_src;
    float scale_y = static_cast<float>(height_dist) / heigth_src;

    hipError_t cudaStatus2 = hipMalloc((void **)&img_src_cuda, heigth_src * width_src * 1 * sizeof(unsigned char *));
    if (cudaStatus2 != hipSuccess) {
        printf("img_src_cuda CudaMalloc failed!");
    }
    hipMemcpy(img_src_cuda, img_src, heigth_src * width_src * sizeof(unsigned char), hipMemcpyHostToDevice);

    nearest_neighbor_interpolate_kernel_launcher(
        stream, height_dist, width_dist, img_dst_cuda, scale_x, scale_y, heigth_src, width_src, img_src_cuda);

    unsigned char *img_dst_cpu = (unsigned char *)malloc(height_dist * width_dist * 1 * sizeof(unsigned char *));
    hipMemcpy(
        img_dst_cpu, img_dst_cuda, height_dist * width_dist * 1 * sizeof(unsigned char *), hipMemcpyDeviceToHost);

    for (int i = 0; i < height_dist * width_dist; i++) {
        if (i % height_dist == 0) {
            printf("\n");
        }
        printf("--%d ", img_dst_cpu[i]);
    }

    hipFree(img_dst_cuda);
    hipFree(img_src_cuda);

    return 0;
}